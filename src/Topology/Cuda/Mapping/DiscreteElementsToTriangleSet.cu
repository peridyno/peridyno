#include "hip/hip_runtime.h"
#include "DiscreteElementsToTriangleSet.h"

namespace dyno
{
	typedef typename ::dyno::TOrientedBox3D<Real> Box3D;

	template<typename TDataType>
	DiscreteElementsToTriangleSet<TDataType>::DiscreteElementsToTriangleSet()
		: TopologyMapping()
	{
		mStandardSphere.loadObjFile(getAssetPath() + "standard/standard_icosahedron.obj");
		mStandardCapsule.loadObjFile(getAssetPath() + "standard/standard_capsule.obj");
	}

	template<typename Triangle>
	__global__ void SetupCubeInstances(
		DArray<Vec3f> vertices,
		DArray<Triangle> indices,
		DArray<Box3D> boxes,
		uint pointOffset,
		uint indexOffset,
		uint cubeOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boxes.size()) return;
		
		int idx = tId;
		Box3D box = boxes[idx];

		Vec3f hx = box.u * box.extent[0];
		Vec3f hy = box.v * box.extent[1];
		Vec3f hz = box.w * box.extent[2];

		Vec3f hyz = hy + hz;
		Vec3f hxy = hx + hy;
		Vec3f hxz = hx + hz;

		Vec3f c = box.center;

		Vec3f v0 = c - hx - hyz;
		Vec3f v1 = c + hx - hyz;
		Vec3f v2 = c + hxz - hy;
		Vec3f v3 = c - hxy + hz;

		Vec3f v4 = c - hxz + hy;
		Vec3f v5 = c + hxy - hz;
		Vec3f v6 = c + hx + hyz;
		Vec3f v7 = c - hx + hyz;

		vertices[pointOffset + idx * 8] = v0;
		vertices[pointOffset + idx * 8 + 1] = v1;
		vertices[pointOffset + idx * 8 + 2] = v2;
		vertices[pointOffset + idx * 8 + 3] = v3;
		vertices[pointOffset + idx * 8 + 4] = v4;
		vertices[pointOffset + idx * 8 + 5] = v5;
		vertices[pointOffset + idx * 8 + 6] = v6;
		vertices[pointOffset + idx * 8 + 7] = v7;

		uint offset = idx * 8 + pointOffset;

		indices[indexOffset + idx * 12] = Triangle(offset + 0, offset + 1, offset + 2);
		indices[indexOffset + idx * 12 + 1] = Triangle(offset + 0, offset + 2, offset + 3);

		indices[indexOffset + idx * 12 + 2] = Triangle(offset + 0, offset + 4, offset + 5);
		indices[indexOffset + idx * 12 + 3] = Triangle(offset + 0, offset + 5, offset + 1);

		indices[indexOffset + idx * 12 + 4] = Triangle(offset + 4, offset + 7, offset + 6);
		indices[indexOffset + idx * 12 + 5] = Triangle(offset + 4, offset + 6, offset + 5);

		indices[indexOffset + idx * 12 + 6] = Triangle(offset + 1, offset + 5, offset + 6);
		indices[indexOffset + idx * 12 + 7] = Triangle(offset + 1, offset + 6, offset + 2);

		indices[indexOffset + idx * 12 + 8] = Triangle(offset + 2, offset + 6, offset + 7);
		indices[indexOffset + idx * 12 + 9] = Triangle(offset + 2, offset + 7, offset + 3);

		indices[indexOffset + idx * 12 + 10] = Triangle(offset + 0, offset + 3, offset + 7);
		indices[indexOffset + idx * 12 + 11] = Triangle(offset + 0, offset + 7, offset + 4);
	}

	template<typename Triangle>
	__global__ void SetupTetInstances(
		DArray<Vec3f> vertices,
		DArray<Triangle> indices,
		DArray<Tet3D> tets,
		uint pointOffset,
		uint indexOffset,
		uint tetOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tets.size()) return;
	
		int idx = tId;
		Tet3D tet = tets[idx];

		Vec3f v0 = tet.v[0];
		Vec3f v1 = tet.v[1];
		Vec3f v2 = tet.v[2];
		Vec3f v3 = tet.v[3];

		vertices[pointOffset + idx * 4] = v0;
		vertices[pointOffset + idx * 4 + 1] = v1;
		vertices[pointOffset + idx * 4 + 2] = v2;
		vertices[pointOffset + idx * 4 + 3] = v3;

		uint offset = idx * 4 + pointOffset;

		indices[indexOffset + idx * 4] = Triangle(offset + 0, offset + 1, offset + 2);
		indices[indexOffset + idx * 4 + 1] = Triangle(offset + 0, offset + 1, offset + 3);
		indices[indexOffset + idx * 4 + 2] = Triangle(offset + 1, offset + 2, offset + 3);
		indices[indexOffset + idx * 4 + 3] = Triangle(offset + 0, offset + 2, offset + 3);
	}

	__global__ void SetupVerticesForSphereInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> sphereVertices,
		DArray<Sphere3D> sphereInstances,
		uint pointOffset,
		uint sphereOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInstances.size() * sphereVertices.size()) return;

		uint instanceId = tId / sphereVertices.size();
		uint vertexId = tId % sphereVertices.size();

		Sphere3D sphere = sphereInstances[instanceId];

		Vec3f v = sphereVertices[vertexId];
		vertices[pointOffset + tId] = sphere.center + sphere.radius * sphere.rotation.rotate(v);
	}

	__global__ void SetupVerticesForMedialConeInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> sphereVertices,
		DArray<MedialCone3D> coneInstances,
		uint pointOffset,
		uint coneOffset
	)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= coneInstances.size() * sphereVertices.size() * 2) return; 

		uint instanceId = tId / (sphereVertices.size() * 2); 
		uint remaining = tId % (sphereVertices.size() * 2);
		uint sphereId = remaining / sphereVertices.size(); 
		uint vertexId = remaining % sphereVertices.size();

		if (instanceId >= coneInstances.size()) return;

		MedialCone3D cone = coneInstances[instanceId];
		Vec3f v = sphereVertices[vertexId];

		vertices[pointOffset + tId] = cone.v[sphereId] + cone.radius[sphereId] * v;
	}

	template<typename Triangle>
	__global__ void SetupIndicesForMedialConeInstances(
		DArray<Triangle> indices,
		DArray<Triangle> sphereIndices,
		DArray<MedialCone3D> coneInstances,
		uint vertexSize,        
		uint vertexDataStartOffset, 
		uint indexOutputStartOffset 
	)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= coneInstances.size() * sphereIndices.size() * 2) return;

		uint instanceId = tId / (sphereIndices.size() * 2);
		uint remaining = tId % (sphereIndices.size() * 2);
		uint sphereId = remaining / sphereIndices.size();
		uint indexId = remaining % sphereIndices.size();

		if (instanceId >= coneInstances.size()) return;


		uint baseVertexOffsetForInstance = vertexDataStartOffset + instanceId * vertexSize * 2;
		uint vertexOffsetForSphere = baseVertexOffsetForInstance + sphereId * vertexSize;

		Triangle tIndex = sphereIndices[indexId];
		indices[indexOutputStartOffset + tId] = Triangle( 
			tIndex[0] + vertexOffsetForSphere,
			tIndex[1] + vertexOffsetForSphere,
			tIndex[2] + vertexOffsetForSphere
		);
	}

	__global__ void SetupVerticesForMedialSlabInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> sphereVertices,
		DArray<MedialSlab3D> slabInstances,
		uint pointOffset,
		uint slabOffset
	)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= slabInstances.size() * sphereVertices.size() * 3) return; 

		uint instanceId = tId / (sphereVertices.size() * 3);
		uint remaining = tId % (sphereVertices.size() * 3);
		uint sphereId = remaining / sphereVertices.size(); 
		uint vertexId = remaining % sphereVertices.size();

		if (instanceId >= slabInstances.size()) return;

		MedialSlab3D slab = slabInstances[instanceId];
		Vec3f v = sphereVertices[vertexId];

		vertices[pointOffset + tId] = slab.v[sphereId] + slab.radius[sphereId] * v;
	}


	template<typename Triangle>
	__global__ void SetupIndicesForMedialSlabInstances(
		DArray<Triangle> indices,
		DArray<Triangle> sphereIndices,
		DArray<MedialSlab3D> slabInstances,
		uint vertexSize,       
		uint vertexDataStartOffset, 
		uint indexOutputStartOffset
	)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= slabInstances.size() * sphereIndices.size() * 3) return;

		uint instanceId = tId / (sphereIndices.size() * 3);
		uint remaining = tId % (sphereIndices.size() * 3);
		uint sphereId = remaining / sphereIndices.size();
		uint indexId = remaining % sphereIndices.size();

		if (instanceId >= slabInstances.size()) return;

		
		uint baseVertexOffsetForInstance = vertexDataStartOffset + instanceId * vertexSize * 3;
		uint vertexOffsetForSphere = baseVertexOffsetForInstance + sphereId * vertexSize;

		Triangle tIndex = sphereIndices[indexId];
		indices[indexOutputStartOffset + tId] = Triangle( 
			tIndex[0] + vertexOffsetForSphere,
			tIndex[1] + vertexOffsetForSphere,
			tIndex[2] + vertexOffsetForSphere
		);
	}
	template<typename Triangle>
	__global__ void SetupIndicesForSphereInstances(
		DArray<Triangle> indices,
		DArray<Triangle> sphereIndices,
		DArray<Sphere3D> sphereInstances,
		uint vertexSize,						
		uint vertexDataStartOffset,         
		uint indexOutputStartOffset         
	)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInstances.size() * sphereIndices.size()) return;

		uint instanceId = tId / sphereIndices.size();
		uint indexId = tId % sphereIndices.size();

		int calculatedVertexOffsetForInstance = vertexDataStartOffset + instanceId * vertexSize; // ʹ����ȷ�Ķ���������ʼƫ����

		Triangle tIndex = sphereIndices[indexId];
		indices[indexOutputStartOffset + tId] = Triangle(tIndex[0] + calculatedVertexOffsetForInstance, tIndex[1] + calculatedVertexOffsetForInstance, tIndex[2] + calculatedVertexOffsetForInstance);
	}

	__global__ void SetupVerticesForCapsuleInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> capsuleVertices,
		DArray<Capsule3D> capsuleInstances,
		uint pointOffset,
		uint capsuleOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= capsuleInstances.size() * capsuleVertices.size()) return;

		uint instanceId = tId / capsuleVertices.size();
		uint vertexId = tId % capsuleVertices.size();

		Capsule3D capsule = capsuleInstances[instanceId];
		float r = capsule.radius;
		float h = capsule.halfLength;
		auto rot = capsule.rotation.toMatrix3x3();
		Vec3f center = capsule.center; 

		Vec3f v = capsuleVertices[vertexId];
		Vec3f orignZ = Vec3f(0, 1, 0);
		Vec3f newZ = Vec3f(0, h, 0);

		if (v.y >= 1)
		{
			vertices[pointOffset + tId] = rot * ((v - orignZ) * r + newZ) + center;
		}
		else if (v.y <= -1) 
		{
			vertices[pointOffset + tId] = rot * ((v + orignZ) * r - newZ) + center;
		}
		else
		{
			vertices[pointOffset + tId] = rot * (v * Vec3f(r, h, r)) + center;
		}	
	}

	template<typename Triangle>
	__global__ void SetupIndicesForCapsuleInstances(
		DArray<Triangle> indices,
		DArray<Triangle> capsuleIndices,
		DArray<Capsule3D> capsuleInstances,
		uint vertexSize,						//vertex size of the instance sphere 
		uint vertexOffset,
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= capsuleInstances.size() * capsuleIndices.size()) return;

		uint instanceId = tId / capsuleIndices.size();
		uint indexId = tId % capsuleIndices.size();

		vertexOffset += instanceId * vertexSize;
		
		Triangle tIndex = capsuleIndices[indexId];
		indices[indexOffset + tId] = Triangle(tIndex[0] + vertexOffset, tIndex[1] + vertexOffset, tIndex[2] + vertexOffset);
	}

	template<typename TDataType>
	bool DiscreteElementsToTriangleSet<TDataType>::apply()
	{
		if (this->outTriangleSet()->isEmpty())
		{
			this->outTriangleSet()->allocate();
		}

		auto inTopo = this->inDiscreteElements()->constDataPtr();

		DArray<Box3D>& boxInGlobal = inTopo->boxesInGlobal();
		DArray<Sphere3D>& sphereInGlobal = inTopo->spheresInGlobal();
		DArray<Tet3D>& tetInGlobal = inTopo->tetsInGlobal();
		DArray<Capsule3D>& capsuleInGlobal = inTopo->capsulesInGlobal();
		DArray<MedialCone3D>& medialConeInGlobal = inTopo->medialConesInGlobal();
		DArray<MedialSlab3D>& medialSlabInGlobal = inTopo->medialSlabsInGlobal();


		ElementOffset elementOffset = inTopo->calculateElementOffset();

		int numOfSpheres = sphereInGlobal.size();
		int numofCaps = capsuleInGlobal.size();
		int numOfBoxes = boxInGlobal.size();
		int numOfTets = tetInGlobal.size();
		int numOfMedialCones = medialConeInGlobal.size();
		int numOfMedialSlabs = medialSlabInGlobal.size();
		
		auto triSet = this->outTriangleSet()->getDataPtr();

		auto& vertices = triSet->getPoints();
		auto& indices = triSet->triangleIndices();

		auto& sphereVertices = mStandardSphere.getPoints();
		auto& sphereIndices = mStandardSphere.triangleIndices();

		auto& capsuleVertices = mStandardCapsule.getPoints();
		auto& capsuleIndices = mStandardCapsule.triangleIndices();
		
		int numOfVertices = 8 * numOfBoxes + 4 * numOfTets + sphereVertices.size() * numOfSpheres + capsuleVertices.size() * numofCaps + 2 * sphereVertices.size() * numOfMedialCones + 3 * sphereVertices.size() * numOfMedialSlabs;
		int numOfTriangles = 12 * numOfBoxes + 4 * numOfTets + sphereIndices.size() * numOfSpheres + capsuleIndices.size() * numofCaps + 2 * sphereIndices.size() * numOfMedialCones + 3 * sphereIndices.size() * numOfMedialSlabs;

		vertices.resize(numOfVertices);
		indices.resize(numOfTriangles);

		uint vertexOffset = 0;
		uint indexOffset = 0;

		//Setup spheres
		cuExecute(numOfSpheres * sphereVertices.size(),
			SetupVerticesForSphereInstances,
			vertices,
			sphereVertices,
			sphereInGlobal,
			vertexOffset,
			elementOffset.sphereIndex());

		cuExecute(numOfSpheres * sphereIndices.size(),
			SetupIndicesForSphereInstances,
			indices,
			sphereIndices,
			sphereInGlobal,
			sphereVertices.size(),
			vertexOffset,
			indexOffset);

		vertexOffset += numOfSpheres * sphereVertices.size();
		indexOffset += numOfSpheres * sphereIndices.size();

		//Setup boxes
		cuExecute(numOfBoxes,
			SetupCubeInstances,
			vertices,
			indices,
			boxInGlobal,
			vertexOffset,
			indexOffset,
			elementOffset.boxIndex());

		vertexOffset += numOfBoxes * 8;
		indexOffset += numOfBoxes * 12;

		//Setup tets
		cuExecute(numOfTets,
			SetupTetInstances,
			vertices,
			indices,
			tetInGlobal,
			vertexOffset,
			indexOffset,
			elementOffset.tetIndex());

		vertexOffset += numOfTets * 4;
		indexOffset += numOfTets * 4;

		cuExecute(numofCaps * capsuleVertices.size(),
			SetupVerticesForCapsuleInstances,
			vertices,
			capsuleVertices,
			capsuleInGlobal,
			vertexOffset,
			elementOffset.capsuleIndex());

		cuExecute(numofCaps * capsuleIndices.size(),
			SetupIndicesForCapsuleInstances,
			indices,
			capsuleIndices,
			capsuleInGlobal,
			capsuleVertices.size(),
			vertexOffset,
			indexOffset);

		vertexOffset += numofCaps * capsuleVertices.size();
		indexOffset += numofCaps * capsuleIndices.size();

		cuExecute(numOfMedialCones * sphereVertices.size() * 2,
			SetupVerticesForMedialConeInstances,
			vertices,
			sphereVertices,
			medialConeInGlobal,
			vertexOffset,
			elementOffset.medialConeIndex());

		cuExecute(numOfMedialCones* sphereIndices.size()* 2	,
			SetupIndicesForMedialConeInstances,
			indices,
			sphereIndices,
			medialConeInGlobal,
			sphereVertices.size(),
			vertexOffset,
			indexOffset);

		vertexOffset += numOfMedialCones * sphereVertices.size() * 2;
		indexOffset += numOfMedialCones * sphereIndices.size() * 2;

		cuExecute(numOfMedialSlabs* sphereVertices.size() * 3,
			SetupVerticesForMedialSlabInstances,
			vertices,
			sphereVertices,
			medialSlabInGlobal,
			vertexOffset,
			elementOffset.medialSlabIndex());

		cuExecute(numOfMedialSlabs* sphereIndices.size() * 3,
			SetupIndicesForMedialSlabInstances,
			indices,
			sphereIndices,
			medialSlabInGlobal,
			sphereVertices.size(),
			vertexOffset,
			indexOffset);

		vertexOffset += numOfMedialSlabs * sphereVertices.size() * 3;
		indexOffset += numOfMedialSlabs * sphereIndices.size() * 3;

		this->outTriangleSet()->getDataPtr()->update();

		return true;
	}

	DEFINE_CLASS(DiscreteElementsToTriangleSet);
}