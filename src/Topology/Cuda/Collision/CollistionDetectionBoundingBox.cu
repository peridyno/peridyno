#include "hip/hip_runtime.h"
#include "CollistionDetectionBoundingBox.h"

#include "Primitive/Primitive3D.h"
#include "Topology/DiscreteElements.h"

namespace dyno
{
	typedef typename ::dyno::TOrientedBox3D<Real> Box3D;

	template<typename TDataType>
	CollistionDetectionBoundingBox<TDataType>::CollistionDetectionBoundingBox()
		: ComputeModule()
	{
	}

	template<typename TDataType>
	CollistionDetectionBoundingBox<TDataType>::~CollistionDetectionBoundingBox()
	{

	}

	template <typename Coord>
	__global__ void CountContactsWithBoundary(
		DArray<Sphere3D> sphere,
		DArray<Box3D> box,
		DArray<Tet3D> tet,
		DArray<Capsule3D> cap,
		DArray<MedialCone3D> medialcone,
		DArray<MedialSlab3D> medialslab,
		DArray<int> count,
		Coord hi,
		Coord lo,
		ElementOffset elementOffset)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= count.size()) return;

		ElementType eleType = elementOffset.checkElementType(pId);

		if (eleType == ET_SPHERE)//sphere
		{
			int cnt = 0;

			Sphere3D sp = sphere[pId - elementOffset.sphereIndex()];

			Real radius = sp.radius;
			Coord center = sp.center;

			if (center.x + radius >= hi.x)
			{
				cnt++;
			}
			if (center.x - radius <= lo.x)
			{
				cnt++;
			}

			if (center.y + radius >= hi.y)
			{
				cnt++;
			}
			if (center.y - radius <= lo.y)
			{
				cnt++;
			}

			if (center.z + radius >= hi.z)
			{
				cnt++;
			}
			if (center.z - radius <= lo.z)
			{
				cnt++;
			}

			count[pId] = cnt;
		}
		else if (eleType == ET_BOX)//box
		{
			//int idx = pId - start_box;
			int cnt = 0;
			//				int start_i;
			Coord center = box[pId - elementOffset.boxIndex()].center;
			Coord u = box[pId - elementOffset.boxIndex()].u;
			Coord v = box[pId - elementOffset.boxIndex()].v;
			Coord w = box[pId - elementOffset.boxIndex()].w;
			Coord extent = box[pId - elementOffset.boxIndex()].extent;
			Point3D p[8];
			p[0] = Point3D(center - u * extent[0] - v * extent[1] - w * extent[2]);
			p[1] = Point3D(center - u * extent[0] - v * extent[1] + w * extent[2]);
			p[2] = Point3D(center - u * extent[0] + v * extent[1] - w * extent[2]);
			p[3] = Point3D(center - u * extent[0] + v * extent[1] + w * extent[2]);
			p[4] = Point3D(center + u * extent[0] - v * extent[1] - w * extent[2]);
			p[5] = Point3D(center + u * extent[0] - v * extent[1] + w * extent[2]);
			p[6] = Point3D(center + u * extent[0] + v * extent[1] - w * extent[2]);
			p[7] = Point3D(center + u * extent[0] + v * extent[1] + w * extent[2]);
			bool c1, c2, c3, c4, c5, c6;
			c1 = c2 = c3 = c4 = c5 = c6 = true;
			for (int i = 0; i < 8; i++)
			{
				Coord pos = p[i].origin;
				if (pos[0] > hi[0] && c1)
				{
					c1 = true;
					cnt++;
				}
				if (pos[1] > hi[1] && c2)
				{
					c2 = true;
					cnt++;
				}
				if (pos[2] > hi[2] && c3)
				{
					c3 = true;
					cnt++;
				}
				if (pos[0] < lo[0] && c4)
				{
					c4 = true;
					cnt++;
				}
				if (pos[1] < lo[1] && c5)
				{
					c5 = true;
					cnt++;
				}
				if (pos[2] < lo[2] && c6)
				{
					c6 = true;
					cnt++;
				}
			}
			count[pId] = cnt;
		}
		else if (eleType == ET_TET) // tets
		{
			int cnt = 0;
			int start_i = count[pId];

			Tet3D tet_i = tet[pId - elementOffset.tetIndex()];

			for (int i = 0; i < 4; i++)
			{
				Coord vertex = tet_i.v[i];
				if (vertex.x >= hi.x)
				{
					cnt++;
				}
				if (vertex.x <= lo.x)
				{
					cnt++;
				}

				if (vertex.y >= hi.y)
				{
					cnt++;
				}
				if (vertex.y <= lo.y)
				{
					cnt++;
				}

				if (vertex.z >= hi.z)
				{
					cnt++;
				}
				if (vertex.z <= lo.z)
				{
					cnt++;
				}
			}

			count[pId] = cnt;
		}
		else if (eleType == ET_CAPSULE)//segments
		{
			int cnt = 0;

			Capsule3D cap_i = cap[pId - elementOffset.capsuleIndex()];

			Coord v0 = cap_i.startPoint();
			Coord v1 = cap_i.endPoint();

			Real radius = cap_i.radius;

			if (v0.x + radius >= hi.x)
			{
				cnt++;
			}
			if (v0.x - radius <= lo.x)
			{
				cnt++;
			}

			if (v0.y + radius >= hi.y)
			{
				cnt++;
			}
			if (v0.y - radius <= lo.y)
			{
				cnt++;
			}

			if (v0.z + radius >= hi.z)
			{
				cnt++;
			}
			if (v0.z - radius <= lo.z)
			{
				cnt++;
			}


			//v1
			if (v1.x + radius >= hi.x)
			{
				cnt++;
			}
			if (v1.x - radius <= lo.x)
			{
				cnt++;
			}

			if (v1.y + radius >= hi.y)
			{
				cnt++;
			}
			if (v1.y - radius <= lo.y)
			{
				cnt++;
			}

			if (v1.z + radius >= hi.z)
			{
				cnt++;
			}
			if (v1.z - radius <= lo.z)
			{
				cnt++;
			}

			count[pId] = cnt;
		}
		else if (eleType == ET_MEDIALCONE) // medialcone
		{
			int cnt = 0;

			MedialCone3D cone_i = medialcone[pId - elementOffset.medialConeIndex()];

			Coord v0 = cone_i.v[0];
			Coord v1 = cone_i.v[1];
			Real radius0 = cone_i.radius[0];
			Real radius1 = cone_i.radius[1];

			// Check v0 with radius0
			if (v0.x + radius0 >= hi.x)
			{
				cnt++;
			}
			if (v0.x - radius0 <= lo.x)
			{
				cnt++;
			}

			if (v0.y + radius0 >= hi.y)
			{
				cnt++;
			}
			if (v0.y - radius0 <= lo.y)
			{
				cnt++;
			}

			if (v0.z + radius0 >= hi.z)
			{
				cnt++;
			}
			if (v0.z - radius0 <= lo.z)
			{
				cnt++;
			}

			// Check v1 with radius1
			if (v1.x + radius1 >= hi.x)
			{
				cnt++;
			}
			if (v1.x - radius1 <= lo.x)
			{
				cnt++;
			}

			if (v1.y + radius1 >= hi.y)
			{
				cnt++;
			}
			if (v1.y - radius1 <= lo.y)
			{
				cnt++;
			}

			if (v1.z + radius1 >= hi.z)
			{
				cnt++;
			}
			if (v1.z - radius1 <= lo.z)
			{
				cnt++;
			}

			count[pId] = cnt;
			}
		else if (eleType == ET_MEDIALSLAB) // medialslab
		{
			int cnt = 0;

			MedialSlab3D slab_i = medialslab[pId - elementOffset.medialSlabIndex()];

			Coord v0 = slab_i.v[0];
			Coord v1 = slab_i.v[1];
			Coord v2 = slab_i.v[2];
			Real radius0 = slab_i.radius[0];
			Real radius1 = slab_i.radius[1];
			Real radius2 = slab_i.radius[2];

			// Check v0 with radius0
			if (v0.x + radius0 >= hi.x)
			{
				cnt++;
			}
			if (v0.x - radius0 <= lo.x)
			{
				cnt++;
			}

			if (v0.y + radius0 >= hi.y)
			{
				cnt++;
			}
			if (v0.y - radius0 <= lo.y)
			{
				cnt++;
			}

			if (v0.z + radius0 >= hi.z)
			{
				cnt++;
			}
			if (v0.z - radius0 <= lo.z)
			{
				cnt++;
			}

			// Check v1 with radius1
			if (v1.x + radius1 >= hi.x)
			{
				cnt++;
			}
			if (v1.x - radius1 <= lo.x)
			{
				cnt++;
			}

			if (v1.y + radius1 >= hi.y)
			{
				cnt++;
			}
			if (v1.y - radius1 <= lo.y)
			{
				cnt++;
			}

			if (v1.z + radius1 >= hi.z)
			{
				cnt++;
			}
			if (v1.z - radius1 <= lo.z)
			{
				cnt++;
			}

			// Check v2 with radius2
			if (v2.x + radius2 >= hi.x)
			{
				cnt++;
			}
			if (v2.x - radius2 <= lo.x)
			{
				cnt++;
			}

			if (v2.y + radius2 >= hi.y)
			{
				cnt++;
			}
			if (v2.y - radius2 <= lo.y)
			{
				cnt++;
			}

			if (v2.z + radius2 >= hi.z)
			{
				cnt++;
			}
			if (v2.z - radius2 <= lo.z)
			{
				cnt++;
			}

			count[pId] = cnt;
			}

	}

	template <typename Coord, typename ContactPair>
	__global__ void SetupContactsWithBoundary(
		DArray<Sphere3D> sphere,
		DArray<Box3D> box,
		DArray<Tet3D> tet,
		DArray<Capsule3D> cap,
		DArray<MedialCone3D> medialcone,
		DArray<MedialSlab3D> medialslab,
		DArray<int> count,
		DArray<ContactPair> nbq,
		DArray<Pair<uint, uint>> mapping,
		Coord hi,
		Coord lo,
		ElementOffset elementOffset)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= count.size()) return;

		ElementType eleType = elementOffset.checkElementType(pId);

		uint rbId = mapping[pId].second;

		if (eleType == ET_SPHERE)//sphere
		{
			int cnt = 0;
			int start_i = count[pId];

			Sphere3D sp = sphere[pId - elementOffset.sphereIndex()];

			Real radius = sp.radius;
			Coord center = sp.center;

			if (center.x + radius >= hi.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
				nbq[cnt + start_i].pos1 = center + Coord(radius, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = center.x + radius - hi.x;
				cnt++;
			}
			if (center.x - radius <= lo.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
				nbq[cnt + start_i].pos1 = center - Coord(radius, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.x - (center.x - radius);
				cnt++;
			}

			if (center.y + radius >= hi.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
				nbq[cnt + start_i].pos1 = center + Coord(0, radius, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = center.y + radius - hi.y;
				cnt++;
			}
			if (center.y - radius <= lo.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
				nbq[cnt + start_i].pos1 = center - Coord(0, radius, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.y - (center.y - radius);
				cnt++;
			}

			if (center.z + radius >= hi.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
				nbq[cnt + start_i].pos1 = center + Coord(0, 0, radius);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = center.z + radius - hi.z;
				cnt++;
			}
			if (center.z - radius <= lo.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
				nbq[cnt + start_i].pos1 = center - Coord(0, 0, radius);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.z - (center.z - radius);
				cnt++;
			}
		}
		else if (eleType == ET_BOX)//box
		{
			//int idx = pId - start_box;
			int cnt = 0;
			int start_i = count[pId];
			Coord center = box[pId - elementOffset.boxIndex()].center;
			Coord u = box[pId - elementOffset.boxIndex()].u;
			Coord v = box[pId - elementOffset.boxIndex()].v;
			Coord w = box[pId - elementOffset.boxIndex()].w;
			Coord extent = box[pId - elementOffset.boxIndex()].extent;
			Point3D p[8];
			p[0] = Point3D(center - u * extent[0] - v * extent[1] - w * extent[2]);
			p[1] = Point3D(center - u * extent[0] - v * extent[1] + w * extent[2]);
			p[2] = Point3D(center - u * extent[0] + v * extent[1] - w * extent[2]);
			p[3] = Point3D(center - u * extent[0] + v * extent[1] + w * extent[2]);
			p[4] = Point3D(center + u * extent[0] - v * extent[1] - w * extent[2]);
			p[5] = Point3D(center + u * extent[0] - v * extent[1] + w * extent[2]);
			p[6] = Point3D(center + u * extent[0] + v * extent[1] - w * extent[2]);
			p[7] = Point3D(center + u * extent[0] + v * extent[1] + w * extent[2]);
			bool c1, c2, c3, c4, c5, c6;
			c1 = c2 = c3 = c4 = c5 = c6 = true;
			for (int i = 0; i < 8; i++)
			{
				Coord pos = p[i].origin;
				if (pos[0] > hi[0] && c1)
				{
					c1 = true;
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = pos[0] - hi[0];
					cnt++;
				}
				if (pos[1] > hi[1] && c2)
				{
					c2 = true;
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = pos[1] - hi[1];
					cnt++;
				}
				if (pos[2] > hi[2] && c3)
				{
					c3 = true;
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = pos[2] - hi[2];
					cnt++;
				}
				if (pos[0] < lo[0] && c4)
				{
					c4 = true;
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo[0] - pos[0];
					cnt++;
				}
				if (pos[1] < lo[1] && c5)
				{
					c5 = true;
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo[1] - pos[1];
					cnt++;
				}
				if (pos[2] < lo[2] && c6)
				{
					c6 = true;
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
					nbq[cnt + start_i].pos1 = pos;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo[2] - pos[2];
					cnt++;
				}

			}
		}
		else if (eleType == ET_TET) // tets
		{
			int cnt = 0;
			int start_i = count[pId];

			Tet3D tet_i = tet[pId - elementOffset.tetIndex()];

			for (int i = 0; i < 4; i++)
			{
				Coord vertex = tet_i.v[i];
				if (vertex.x >= hi.x)
				{
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
					nbq[cnt + start_i].pos1 = vertex;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = vertex.x - hi.x;
					cnt++;
				}
				if (vertex.x <= lo.x)
				{
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
					nbq[cnt + start_i].pos1 = vertex;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo.x - (vertex.x);
					cnt++;
				}

				if (vertex.y >= hi.y)
				{
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
					nbq[cnt + start_i].pos1 = vertex;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = vertex.y - hi.y;
					cnt++;
				}
				if (vertex.y <= lo.y)
				{
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
					nbq[cnt + start_i].pos1 = vertex;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo.y - (vertex.y);
					cnt++;
				}

				if (vertex.z >= hi.z)
				{
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
					nbq[cnt + start_i].pos1 = vertex;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = vertex.z - hi.z;
					cnt++;
				}
				if (vertex.z <= lo.z)
				{
					nbq[cnt + start_i].bodyId1 = rbId;
					nbq[cnt + start_i].bodyId2 = -1;
					nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
					nbq[cnt + start_i].pos1 = vertex;
					nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
					nbq[cnt + start_i].interpenetration = lo.z - (vertex.z);
					cnt++;
				}
			}
		}
		else if (eleType == ET_CAPSULE)
		{
			int cnt = 0;
			int start_i = count[pId];

			Capsule3D cap_i = cap[pId - elementOffset.capsuleIndex()];

			Coord v0 = cap_i.startPoint();
			Coord v1 = cap_i.endPoint();

			Real radius = cap_i.radius;

			//v0
			if (v0.x + radius >= hi.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
				nbq[cnt + start_i].pos1 = v0 + Coord(radius, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.x + radius - hi.x;
				cnt++;
			}
			if (v0.x - radius <= lo.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
				nbq[cnt + start_i].pos1 = v0 - Coord(radius, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.x - (v0.x - radius);
				cnt++;
			}

			if (v0.y + radius >= hi.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
				nbq[cnt + start_i].pos1 = v0 + Coord(0, radius, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.y + radius - hi.y;
				cnt++;
			}
			if (v0.y - radius <= lo.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
				nbq[cnt + start_i].pos1 = v0 - Coord(0, radius, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.y - (v0.y - radius);
				cnt++;
			}

			if (v0.z + radius >= hi.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
				nbq[cnt + start_i].pos1 = v0 + Coord(0, 0, radius);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.z + radius - hi.z;
				cnt++;
			}
			if (v0.z - radius <= lo.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
				nbq[cnt + start_i].pos1 = v0 - Coord(0, 0, radius);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.z - (v0.z - radius);
				cnt++;
			}

			//v1
			if (v1.x + radius >= hi.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
				nbq[cnt + start_i].pos1 = v1 + Coord(radius, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.x + radius - hi.x;
				cnt++;
			}
			if (v1.x - radius <= lo.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
				nbq[cnt + start_i].pos1 = v1 - Coord(radius, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.x - (v1.x - radius);
				cnt++;
			}

			if (v1.y + radius >= hi.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
				nbq[cnt + start_i].pos1 = v1 + Coord(0, radius, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.y + radius - hi.y;
				cnt++;
			}
			if (v1.y - radius <= lo.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
				nbq[cnt + start_i].pos1 = v1 - Coord(0, radius, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.y - (v1.y - radius);
				cnt++;
			}

			if (v1.z + radius >= hi.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
				nbq[cnt + start_i].pos1 = v1 + Coord(0, 0, radius);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.z + radius - hi.z;
				cnt++;
			}
			if (v1.z - radius <= lo.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
				nbq[cnt + start_i].pos1 = v1 - Coord(0, 0, radius);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.z - (v1.z - radius);
				cnt++;
			}
		}
		else if (eleType == ET_MEDIALCONE)
		{
			int cnt = 0;
			int start_i = count[pId];

			MedialCone3D cone_i = medialcone[pId - elementOffset.medialConeIndex()];

			Coord v0 = cone_i.v[0];
			Coord v1 = cone_i.v[1];
			Real radius0 = cone_i.radius[0];
			Real radius1 = cone_i.radius[1];

			// v0 boundary contacts
			if (v0.x + radius0 >= hi.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
				nbq[cnt + start_i].pos1 = v0 + Coord(radius0, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.x + radius0 - hi.x;
				cnt++;
			}
			if (v0.x - radius0 <= lo.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
				nbq[cnt + start_i].pos1 = v0 - Coord(radius0, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.x - (v0.x - radius0);
				cnt++;
			}

			if (v0.y + radius0 >= hi.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
				nbq[cnt + start_i].pos1 = v0 + Coord(0, radius0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.y + radius0 - hi.y;
				cnt++;
			}
			if (v0.y - radius0 <= lo.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
				nbq[cnt + start_i].pos1 = v0 - Coord(0, radius0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.y - (v0.y - radius0);
				cnt++;
			}

			if (v0.z + radius0 >= hi.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
				nbq[cnt + start_i].pos1 = v0 + Coord(0, 0, radius0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.z + radius0 - hi.z;
				cnt++;
			}
			if (v0.z - radius0 <= lo.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
				nbq[cnt + start_i].pos1 = v0 - Coord(0, 0, radius0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.z - (v0.z - radius0);
				cnt++;
			}

			// v1 boundary contacts
			if (v1.x + radius1 >= hi.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
				nbq[cnt + start_i].pos1 = v1 + Coord(radius1, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.x + radius1 - hi.x;
				cnt++;
			}
			if (v1.x - radius1 <= lo.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
				nbq[cnt + start_i].pos1 = v1 - Coord(radius1, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.x - (v1.x - radius1);
				cnt++;
			}

			if (v1.y + radius1 >= hi.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
				nbq[cnt + start_i].pos1 = v1 + Coord(0, radius1, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.y + radius1 - hi.y;
				cnt++;
			}
			if (v1.y - radius1 <= lo.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
				nbq[cnt + start_i].pos1 = v1 - Coord(0, radius1, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.y - (v1.y - radius1);
				cnt++;
			}

			if (v1.z + radius1 >= hi.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
				nbq[cnt + start_i].pos1 = v1 + Coord(0, 0, radius1);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.z + radius1 - hi.z;
				cnt++;
			}
			if (v1.z - radius1 <= lo.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
				nbq[cnt + start_i].pos1 = v1 - Coord(0, 0, radius1);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.z - (v1.z - radius1);
				cnt++;
			}
			}
		else if (eleType == ET_MEDIALSLAB)
		{
			int cnt = 0;
			int start_i = count[pId];

			MedialSlab3D slab_i = medialslab[pId - elementOffset.medialSlabIndex()];

			Coord v0 = slab_i.v[0];
			Coord v1 = slab_i.v[1];
			Coord v2 = slab_i.v[2];
			Real radius0 = slab_i.radius[0];
			Real radius1 = slab_i.radius[1];
			Real radius2 = slab_i.radius[2];

			// v0 boundary contacts
			if (v0.x + radius0 >= hi.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
				nbq[cnt + start_i].pos1 = v0 + Coord(radius0, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.x + radius0 - hi.x;
				cnt++;
			}
			if (v0.x - radius0 <= lo.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
				nbq[cnt + start_i].pos1 = v0 - Coord(radius0, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.x - (v0.x - radius0);
				cnt++;
			}

			if (v0.y + radius0 >= hi.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
				nbq[cnt + start_i].pos1 = v0 + Coord(0, radius0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.y + radius0 - hi.y;
				cnt++;
			}
			if (v0.y - radius0 <= lo.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
				nbq[cnt + start_i].pos1 = v0 - Coord(0, radius0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.y - (v0.y - radius0);
				cnt++;
			}

			if (v0.z + radius0 >= hi.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
				nbq[cnt + start_i].pos1 = v0 + Coord(0, 0, radius0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v0.z + radius0 - hi.z;
				cnt++;
			}
			if (v0.z - radius0 <= lo.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
				nbq[cnt + start_i].pos1 = v0 - Coord(0, 0, radius0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.z - (v0.z - radius0);
				cnt++;
			}

			// v1 boundary contacts
			if (v1.x + radius1 >= hi.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
				nbq[cnt + start_i].pos1 = v1 + Coord(radius1, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.x + radius1 - hi.x;
				cnt++;
			}
			if (v1.x - radius1 <= lo.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
				nbq[cnt + start_i].pos1 = v1 - Coord(radius1, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.x - (v1.x - radius1);
				cnt++;
			}

			if (v1.y + radius1 >= hi.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
				nbq[cnt + start_i].pos1 = v1 + Coord(0, radius1, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.y + radius1 - hi.y;
				cnt++;
			}
			if (v1.y - radius1 <= lo.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
				nbq[cnt + start_i].pos1 = v1 - Coord(0, radius1, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.y - (v1.y - radius1);
				cnt++;
			}

			if (v1.z + radius1 >= hi.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
				nbq[cnt + start_i].pos1 = v1 + Coord(0, 0, radius1);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v1.z + radius1 - hi.z;
				cnt++;
			}
			if (v1.z - radius1 <= lo.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
				nbq[cnt + start_i].pos1 = v1 - Coord(0, 0, radius1);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.z - (v1.z - radius1);
				cnt++;
			}

			// v2 boundary contacts
			if (v2.x + radius2 >= hi.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(-1, 0, 0);
				nbq[cnt + start_i].pos1 = v2 + Coord(radius2, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v2.x + radius2 - hi.x;
				cnt++;
			}
			if (v2.x - radius2 <= lo.x)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(1, 0, 0);
				nbq[cnt + start_i].pos1 = v2 - Coord(radius2, 0, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.x - (v2.x - radius2);
				cnt++;
			}

			if (v2.y + radius2 >= hi.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, -1, 0);
				nbq[cnt + start_i].pos1 = v2 + Coord(0, radius2, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v2.y + radius2 - hi.y;
				cnt++;
			}
			if (v2.y - radius2 <= lo.y)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 1, 0);
				nbq[cnt + start_i].pos1 = v2 - Coord(0, radius2, 0);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.y - (v2.y - radius2);
				cnt++;
			}

			if (v2.z + radius2 >= hi.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, -1);
				nbq[cnt + start_i].pos1 = v2 + Coord(0, 0, radius2);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = v2.z + radius2 - hi.z;
				cnt++;
			}
			if (v2.z - radius2 <= lo.z)
			{
				nbq[cnt + start_i].bodyId1 = rbId;
				nbq[cnt + start_i].bodyId2 = -1;
				nbq[cnt + start_i].normal1 = Coord(0, 0, 1);
				nbq[cnt + start_i].pos1 = v2 - Coord(0, 0, radius2);
				nbq[cnt + start_i].contactType = ContactType::CT_BOUDNARY;
				nbq[cnt + start_i].interpenetration = lo.z - (v2.z - radius2);
				cnt++;
			}
			}
	}

	template<typename TDataType>
	void CollistionDetectionBoundingBox<TDataType>::compute()
	{
		int sum = 0;

		auto upperBound = this->varUpperBound()->getData();
		auto lowerBound = this->varLowerBound()->getData();

		auto discreteSet = this->inDiscreteElements()->getDataPtr();
		uint totalSize = discreteSet->totalSize();

		DArray<Box3D>& boxInGlobal = discreteSet->boxesInGlobal();
		DArray<Sphere3D>& sphereInGlobal = discreteSet->spheresInGlobal();
		DArray<Tet3D>& tetInGlobal = discreteSet->tetsInGlobal();
		DArray<Capsule3D>& capsuleInGlobal = discreteSet->capsulesInGlobal();
		DArray<MedialCone3D>& medialConeInGlobal = discreteSet->medialConesInGlobal();
		DArray<MedialSlab3D>& medialSlabInGlobal = discreteSet->medialSlabsInGlobal();

		ElementOffset offset = discreteSet->calculateElementOffset();

		mBoundaryContactCounter.resize(discreteSet->totalSize());
		mBoundaryContactCounter.reset();
		if (discreteSet->totalSize() > 0)
		{
			cuExecute(totalSize,
				CountContactsWithBoundary,
				sphereInGlobal,
				boxInGlobal,
				tetInGlobal,
				capsuleInGlobal,
				medialConeInGlobal,
				medialSlabInGlobal,
				mBoundaryContactCounter,
				upperBound,
				lowerBound,
				offset);

			sum += mReduce.accumulate(mBoundaryContactCounter.begin(), mBoundaryContactCounter.size());
			mScan.exclusive(mBoundaryContactCounter, true);

			this->outContacts()->resize(sum);

			if (sum > 0) {
				cuExecute(totalSize,
					SetupContactsWithBoundary,
					sphereInGlobal,
					boxInGlobal,
					tetInGlobal,
					capsuleInGlobal,
					medialConeInGlobal,
					medialSlabInGlobal,
					mBoundaryContactCounter,
					this->outContacts()->getData(),
					discreteSet->shape2RigidBodyMapping(),
					upperBound,
					lowerBound,
					offset);
			}
		}
		else
			this->outContacts()->resize(0);
	}

	DEFINE_CLASS(CollistionDetectionBoundingBox);
}