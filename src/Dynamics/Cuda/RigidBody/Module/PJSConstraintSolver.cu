#include "hip/hip_runtime.h"
#include "PJSConstraintSolver.h"
#include "SharedFuncsForRigidBody.h"
//#define USE_RELAXATION
#define FILE_NAME "D:/Work Code/peridyno/Data/v2.txt"
namespace dyno
{
	IMPLEMENT_TCLASS(PJSConstraintSolver, TDataType)

	template<typename TDataType>
	PJSConstraintSolver<TDataType>::PJSConstraintSolver()
		:ConstraintModule()
	{
		this->inContacts()->tagOptional(true);
	}

	template<typename TDataType>
	PJSConstraintSolver<TDataType>::~PJSConstraintSolver()
	{
	}

	template<typename TDataType>
	void PJSConstraintSolver<TDataType>::initializeRelaxation()
	{
		int constraint_size = 0;
		int contact_size = this->inContacts()->size();

		auto topo = this->inDiscreteElements()->constDataPtr();

		int ballAndSocketJoint_size = topo->ballAndSocketJoints().size();
		int sliderJoint_size = topo->sliderJoints().size();
		int hingeJoint_size = topo->hingeJoints().size();
		int fixedJoint_size = topo->fixedJoints().size();
		int pointJoint_size = topo->pointJoints().size();

		if (this->varFrictionEnabled()->getData())
		{
			constraint_size += 3 * contact_size;
		}
		else
		{
			constraint_size = contact_size;
		}

		if (ballAndSocketJoint_size != 0)
		{
			constraint_size += 3 * ballAndSocketJoint_size;
		}

		if (sliderJoint_size != 0)
		{
			constraint_size += 8 * sliderJoint_size;
		}

		if (hingeJoint_size != 0)
		{
			constraint_size += 8 * hingeJoint_size;
		}

		if (fixedJoint_size != 0)
		{
			constraint_size += 6 * fixedJoint_size;
		}

		if (pointJoint_size != 0)
		{
			constraint_size += 3 * pointJoint_size;
		}

		if (constraint_size == 0)
		{
			return;
		}

		mVelocityConstraints.resize(constraint_size);

		if (contact_size != 0)
		{
			auto& contacts = this->inContacts()->getData();
			setUpContactAndFrictionConstraints(
				mVelocityConstraints,
				mContactsInLocalFrame,
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData(),
				this->varFrictionEnabled()->getData()
			);
		}

		if (ballAndSocketJoint_size != 0)
		{
			auto& joints = topo->ballAndSocketJoints();
			int begin_index = contact_size;

			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}

			setUpBallAndSocketJointConstraints(
				mVelocityConstraints,
				joints,
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData(),
				begin_index
			);
		}

		if (sliderJoint_size != 0)
		{
			auto& joints = topo->sliderJoints();
			int begin_index = contact_size;

			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}
			begin_index += 3 * ballAndSocketJoint_size;
			setUpSliderJointConstraints(
				mVelocityConstraints,
				joints,
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData(),
				this->inQuaternion()->getData(),
				begin_index
			);
		}

		if (hingeJoint_size != 0)
		{
			auto& joints = topo->hingeJoints();
			int begin_index = contact_size + 3 * ballAndSocketJoint_size + 8 * sliderJoint_size;
			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}
			setUpHingeJointConstraints(
				mVelocityConstraints,
				joints,
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData(),
				this->inQuaternion()->getData(),
				begin_index
			);
		}

		if (fixedJoint_size != 0)
		{
			auto& joints = topo->fixedJoints();
			int begin_index = contact_size + 3 * ballAndSocketJoint_size + 8 * sliderJoint_size + 8 * hingeJoint_size;
			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}
			setUpFixedJointConstraints(
				mVelocityConstraints,
				joints,
				this->inRotationMatrix()->getData(),
				this->inQuaternion()->getData(),
				begin_index
			);
		}

		if (pointJoint_size != 0)
		{
			auto& joints = topo->pointJoints();
			int begin_index = contact_size + 3 * ballAndSocketJoint_size + 8 * sliderJoint_size + 8 * hingeJoint_size + 6 * fixedJoint_size;
			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}
			setUpPointJointConstraints(
				mVelocityConstraints,
				joints,
				this->inCenter()->getData(),
				begin_index
			);
		}

		auto sizeOfRigids = this->inCenter()->size();
		mContactNumber.resize(sizeOfRigids);

		mJ.resize(4 * constraint_size);
		mB.resize(4 * constraint_size);
		mK_1.resize(constraint_size);
		mK_2.resize(constraint_size);
		mK_3.resize(constraint_size);
		mEta.resize(constraint_size);

		mJ.reset();
		mB.reset();
		mK_1.reset();
		mK_2.reset();
		mK_3.reset();
		mEta.reset();

		calculateJacobianMatrix(
			mJ,
			mB,
			this->inCenter()->getData(),
			this->inInertia()->getData(),
			this->inMass()->getData(),
			this->inRotationMatrix()->getData(),
			mVelocityConstraints
		);

		calculateK(
			mVelocityConstraints,
			mJ,
			mB,
			this->inCenter()->getData(),
			this->inInertia()->getData(),
			this->inMass()->getData(),
			mK_1,
			mK_2,
			mK_3
		);

		calculateEtaVectorForRelaxation(
			mEta,
			mJ,
			this->inVelocity()->getData(),
			this->inAngularVelocity()->getData(),
			mVelocityConstraints
		);


	}

	template<typename TDataType>
	void PJSConstraintSolver<TDataType>::initializeJacobian(Real dt)
	{
		int constraint_size = 0;
		int contact_size = this->inContacts()->size();

		auto topo = this->inDiscreteElements()->constDataPtr();

		int ballAndSocketJoint_size = topo->ballAndSocketJoints().size();
		int sliderJoint_size = topo->sliderJoints().size();
		int hingeJoint_size = topo->hingeJoints().size();
		int fixedJoint_size = topo->fixedJoints().size();
		int pointJoint_size = topo->pointJoints().size();

		if (this->varFrictionEnabled()->getData())
		{
			constraint_size += 3 * contact_size;
		}
		else
		{
			constraint_size = contact_size;
		}

		if (ballAndSocketJoint_size != 0)
		{
			constraint_size += 3 * ballAndSocketJoint_size;
		}

		if (sliderJoint_size != 0)
		{
			constraint_size += 8 * sliderJoint_size;
		}

		if (hingeJoint_size != 0)
		{
			constraint_size += 8 * hingeJoint_size;
		}

		if (fixedJoint_size != 0)
		{
			constraint_size += 6 * fixedJoint_size;
		}

		if (pointJoint_size != 0)
		{
			constraint_size += 3 * pointJoint_size;
		}

		if (constraint_size == 0)
		{
			return;
		}

		mVelocityConstraints.resize(constraint_size);

		if (contact_size != 0)
		{
			if (mContactsInLocalFrame.size() != this->inContacts()->size()) {
				mContactsInLocalFrame.resize(this->inContacts()->size());
			}

			setUpContactsInLocalFrame(
				mContactsInLocalFrame,
				this->inContacts()->getData(),
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData()
			);

			auto& contacts = this->inContacts()->getData();
			setUpContactAndFrictionConstraints(
				mVelocityConstraints,
				mContactsInLocalFrame,
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData(),
				this->varFrictionEnabled()->getData()
			);
		}

		if (ballAndSocketJoint_size != 0)
		{
			auto& joints = topo->ballAndSocketJoints();
			int begin_index = contact_size;

			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}

			setUpBallAndSocketJointConstraints(
				mVelocityConstraints,
				joints,
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData(),
				begin_index
			);
		}

		if (sliderJoint_size != 0)
		{
			auto& joints = topo->sliderJoints();
			int begin_index = contact_size;

			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}
			begin_index += 3 * ballAndSocketJoint_size;
			setUpSliderJointConstraints(
				mVelocityConstraints,
				joints,
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData(),
				this->inQuaternion()->getData(),
				begin_index
			);
		}

		if (hingeJoint_size != 0)
		{
			auto& joints = topo->hingeJoints();
			int begin_index = contact_size + 3 * ballAndSocketJoint_size + 8 * sliderJoint_size;
			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}
			setUpHingeJointConstraints(
				mVelocityConstraints,
				joints,
				this->inCenter()->getData(),
				this->inRotationMatrix()->getData(),
				this->inQuaternion()->getData(),
				begin_index
			);
		}

		if (fixedJoint_size != 0)
		{
			auto& joints = topo->fixedJoints();
			int begin_index = contact_size + 3 * ballAndSocketJoint_size + 8 * sliderJoint_size + 8 * hingeJoint_size;
			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}
			setUpFixedJointConstraints(
				mVelocityConstraints,
				joints,
				this->inRotationMatrix()->getData(),
				this->inQuaternion()->getData(),
				begin_index
			);;
		}

		if (pointJoint_size != 0)
		{
			auto& joints = topo->pointJoints();
			int begin_index = contact_size + 3 * ballAndSocketJoint_size + 8 * sliderJoint_size + 8 * hingeJoint_size + 6 * fixedJoint_size;
			if (this->varFrictionEnabled()->getData())
			{
				begin_index += 2 * contact_size;
			}
			setUpPointJointConstraints(
				mVelocityConstraints,
				joints,
				this->inCenter()->getData(),
				begin_index
			);
		}

		auto sizeOfRigids = this->inCenter()->size();
		mContactNumber.resize(sizeOfRigids);

		mJ.resize(4 * constraint_size);
		mB.resize(4 * constraint_size);
		mK_1.resize(constraint_size);
		mK_2.resize(constraint_size);
		mK_3.resize(constraint_size);
		mEta.resize(constraint_size);
		mLambda.resize(constraint_size);
		

		

		mJ.reset();
		mB.reset();
		mK_1.reset();
		mK_2.reset();
		mK_3.reset();
		mEta.reset();
		mLambda.reset();
		

		mContactNumber.reset();

		calculateJacobianMatrix(
			mJ,
			mB,
			this->inCenter()->getData(),
			this->inInertia()->getData(),
			this->inMass()->getData(),
			this->inRotationMatrix()->getData(),
			mVelocityConstraints
		);

		mErrors.resize(constraint_size);
		mErrors.reset();

		calculateEtaVectorForPJSBaumgarte(
			mEta,
			mJ,
			this->inVelocity()->getData(),
			this->inAngularVelocity()->getData(),
			this->inCenter()->getData(),
			this->inQuaternion()->getData(),
			mVelocityConstraints,
			mErrors,
			this->varSlop()->getValue(),
			this->varBaumgarteRate()->getValue(),
			1,
			dt
		);
		
		calculateK(
			mVelocityConstraints,
			mJ,
			mB,
			this->inCenter()->getData(),
			this->inInertia()->getData(),
			this->inMass()->getData(),
			mK_1,
			mK_2,
			mK_3
		);

		if (contact_size != 0)
		{
			calculateContactPoints(
				this->inContacts()->getData(),
				mContactNumber);
		}
	}

	template<typename TDataType>
	void PJSConstraintSolver<TDataType>::constrain()
	{
		uint bodyNum = this->inCenter()->size();

		auto topo = this->inDiscreteElements()->constDataPtr();

		mImpulseC.resize(bodyNum * 2);
		mImpulseExt.resize(bodyNum * 2);
		mImpulseC.reset();
		mImpulseExt.reset();

		Real dt = this->inTimeStep()->getData();

		if (this->varGravityEnabled()->getValue())
		{
			setUpGravity(
				mImpulseExt,
				this->varGravityValue()->getValue(),
				dt
			);
		}


		updateVelocity(
			this->inAttribute()->getData(),
			this->inVelocity()->getData(),
			this->inAngularVelocity()->getData(),
			mImpulseExt,
			this->varLinearDamping()->getValue(),
			this->varAngularDamping()->getValue(),
			dt
		);

		if (!this->inContacts()->isEmpty() || topo->totalJointSize() > 0)
		{
			int contact_size = this->inContacts()->size();
			initializeJacobian(dt);
			errors.push_back(checkOutErrors(mErrors));
			int constraint_size = mVelocityConstraints.size();

			std::vector<std::pair<int, int>> edges;
			
			constraintsMappingToEdges(mVelocityConstraints, edges);

			/*if (!mGraphColoring.isGraphInitialized()) {
				mGraphColoring.initializeGraph(
					bodyNum,
					edges
				);
				mGraphColoring.performInitialColoring();
				std::cout << "Graph Coloring Initialized with " << mGraphColoring.getNumColors() << " colors." << std::endl;
			}*/

			for (int i = 0; i < this->varIterationNumberForVelocitySolver()->getValue(); i++)
			{
				JacobiIteration(
					mLambda,
					mImpulseC,
					mJ,
					mB,
					mEta,
					mVelocityConstraints,
					mContactNumber,
					mK_1,
					mK_2,
					mK_3,
					this->inMass()->getData(),
					this->inFrictionCoefficients()->getData(),
					this->varFrictionCoefficient()->getData(),
					this->varGravityValue()->getData(),
					dt
				);
			}

			Real norm = checkOutError(
				mJ,
				mImpulseC,
				mVelocityConstraints,
				mEta
			);

			
			errors.push_back(norm);

			updateVelocity(
				this->inAttribute()->getData(),
				this->inVelocity()->getData(),
				this->inAngularVelocity()->getData(),
				mImpulseC,
				this->varLinearDamping()->getValue(),
				this->varAngularDamping()->getValue(),
				dt
			);

			updateGesture(
				this->inAttribute()->getData(),
				this->inCenter()->getData(),
				this->inQuaternion()->getData(),
				this->inRotationMatrix()->getData(),
				this->inInertia()->getData(),
				this->inVelocity()->getData(),
				this->inAngularVelocity()->getData(),
				this->inInitialInertia()->getData(),
				dt
			);

			#ifdef USE_RELAXATION
				// Relaxation Step
				initializeRelaxation();

				for (int i = 0; i < 30; i++)
				{
					JacobiIteration(
						mLambda,
						mImpulseC,
						mJ,
						mB,
						mEta,
						mVelocityConstraints,
						mContactNumber,
						mK_1,
						mK_2,
						mK_3,
						this->inMass()->getData(),
						this->varFrictionCoefficient()->getData(),
						this->varGravityValue()->getData(),
						dt
					);
				}

				updateVelocity(
					this->inAttribute()->getData(),
					this->inVelocity()->getData(),
					this->inAngularVelocity()->getData(),
					mImpulseC,
					0.0f,
					0.0f,
					dt
				);
			#else
			
			#endif

		}
		else
		{
			updateGesture(
				this->inAttribute()->getData(),
				this->inCenter()->getData(),
				this->inQuaternion()->getData(),
				this->inRotationMatrix()->getData(),
				this->inInertia()->getData(),
				this->inVelocity()->getData(),
				this->inAngularVelocity()->getData(),
				this->inInitialInertia()->getData(),
				dt
			);
		}


		
		
	}

	DEFINE_CLASS(PJSConstraintSolver);
}